#include "hip/hip_runtime.h"
#include "My_GPU.h"

__global__ void im2col_kernel(const float *image, float *data_col, int height_in, int width_in, int height_kernel, int width_kernel, int height_out, int width_out, int stride, int pad_h, int pad_w, int channel_in)
{
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= hw_out * hw_kernel * channel_in)
        return;

    int c = idx / (hw_out * hw_kernel);
    idx %= hw_out * hw_kernel;
    int i = idx / hw_kernel;
    int j = idx % hw_kernel;

    int step_h = i / width_out;
    int step_w = i % width_out;
    int start_idx = step_h * width_in * stride + step_w * stride;

    int cur_col = start_idx % width_in + j % width_kernel - pad_w;
    int cur_row = start_idx / width_in + j / width_kernel - pad_h;

    if (cur_col < 0 || cur_col >= width_in || cur_row < 0 || cur_row >= height_in)
    {
        data_col[idx] = 0;
    }
    else
    {
        int pick_idx = cur_row * width_in + cur_col;
        data_col[idx] = image[c * hw_in + pick_idx];
    }
}
